#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../inc/argument_utils.h"

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

typedef int64_t int_t;
typedef double real_t;

int_t
    M,
    N,
    max_iteration,
    snapshot_frequency,
    size;

real_t
    *h_temp[2] = { NULL, NULL },
    *h_thermal_diffusivity,
    // TODO 1: Declare device side pointers to store host-side data.
    *d_temp,
    *d_temp_next,
    *d_thermal_diffusivity,
    dt;

#define T(x,y)                      h_temp[0][(y) * (N + 2) + (x)]
#define T_next(x,y)                 h_temp[1][((y) * (N + 2) + (x))]
#define THERMAL_DIFFUSIVITY(x,y)    h_thermal_diffusivity[(y) * (N + 2) + (x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step (const int_t N, const int_t M, real_t *d_temp, real_t *d_temp_next, const real_t *d_thermal_diffusivity, const real_t dt);
__device__ void boundary_condition(const int x, const int y, const int_t N, const int_t M, real_t *d_temp);
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


void
swap ( real_t* &m1, real_t* &m2)
{
    real_t* tmp;
    tmp = m1;
    m1 = m2;
    m2 = tmp;
}


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    M = options->M;
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    dim3 threadBlockDims(32,32);
    dim3 gridDims((N+2)/32+1, (M+2)/32+1);

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO 6: Launch the time_step-kernel.
        time_step <<< gridDims, threadBlockDims >>> (N, M, d_temp, d_temp_next, d_thermal_diffusivity, dt);

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO 8: Copy data from device to host.
            hipMemcpy(h_temp[0], d_temp, size, hipMemcpyDeviceToHost);
            domain_save ( iteration );
        }

        // TODO 7: Swap device pointers.
        // swap( &h_temp[0], &h_temp[1] );
        std::swap(d_temp, d_temp_next);
    }

    gettimeofday ( &t_end, NULL );
    printf ( "Total elapsed time: %lf seconds\n",
            WALLTIME(t_end) - WALLTIME(t_start)
            );


    domain_finalize();

    exit ( EXIT_SUCCESS );
}


// TODO 4: Make time_step() a CUDA kernel
//         where one thread is responsible for one grid point.

__global__ void time_step (const int_t N, const int_t M, real_t *d_temp, real_t *d_temp_next, const real_t *d_thermal_diffusivity, const real_t dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 

    if (idx >= N+2 || idy >= M+2){return;}

    boundary_condition(idx, idy, N, M, d_temp);

    __syncthreads();

    if ((1 <= idx && idx <= N) && (1 <= idy && idy <= M)){
        real_t c, t, b, l, r, K, new_value;
        //(y) * (N + 2) + (x)
        c = d_temp[idy * (N + 2) + idx];

        t = d_temp[idy * (N + 2) + idx - 1];
        b = d_temp[idy * (N + 2) + idx + 1];
        l = d_temp[(idy-1) * (N + 2) + idx];
        r = d_temp[(idy+1) * (N + 2) + idx];
        K = d_thermal_diffusivity[idy * (N + 2) + idx];

        new_value = c + K * dt * ((l - 2 * c + r) + (b - 2 * c + t));

        d_temp_next[idy * (N + 2) + idx] = new_value;   
    }


    

    // real_t c, t, b, l, r, K, new_value;
    // for ( int_t y = 1; y <= M; y++ )
    // {
    //     for ( int_t x = 1; x <= N; x++ )
    //     {
    //         c = T(x, y);

    //         t = T(x - 1, y);
    //         b = T(x + 1, y);
    //         l = T(x, y - 1);
    //         r = T(x, y + 1);
    //         K = THERMAL_DIFFUSIVITY(x, y);

    //         new_value = c + K * dt * ((l - 2 * c + r) + (b - 2 * c + t));

    //         T_next(x, y) = new_value;
    //     }
    // }
}


// TODO 5: Make boundary_condition() a device function and
//         call it from the time_step-kernel.
//         Chose appropriate threads to set the boundary values.


__device__ void boundary_condition (const int x, const int y, const int_t N, const int_t M, real_t *d_temp)
{
    if(x == 0 && (1 <= y && y <= M)){d_temp[y * (N + 2)] = d_temp[y * (N + 2) + 2];}
    else if(x == N+1 && (1 <= y && y <= M)){d_temp[y * (N + 2) + N+1] = d_temp[y * (N + 2) + N-1];}
    else if(y == 0 && (1 <= x && x <= N)){d_temp[x] = d_temp[2 * (N + 2) + x];}
    else if(y == M+1 && (1 <= x && x <= N)){d_temp[(M+1) * (N + 2) + x] = d_temp[(M-1) * (N + 2) + x];}
}


void
domain_init ( void )
{
    size = (M+2)*(N+2) * sizeof(real_t);
    h_temp[0] = (real_t*) malloc (size);
    h_temp[1] = (real_t*) malloc (size);
    h_thermal_diffusivity = (real_t*) malloc (size);

    // TODO 2: Allocate device memory.
    hipMalloc(&d_temp, size);
    hipMalloc(&d_temp_next, size);
    hipMalloc(&d_thermal_diffusivity, size);


    dt = 0.1;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            real_t temperature = 30 + 30 * sin((x + y) / 20.0);
            real_t diffusivity = 0.05 + (30 + 30 * sin((N - x + y) / 20.0)) / 605.0;

            h_temp[0][ y*(N+2) + x ] = temperature;
            h_temp[1][ y*(N+2) + x ] = temperature;
            h_thermal_diffusivity[ y*(N+2) + x ] = diffusivity;
        }
    }

    // TODO 3: Copy data from host to device.
    hipMemcpy(d_temp, h_temp[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp_next, h_temp[1], size, hipMemcpyHostToDevice);
    hipMemcpy(d_thermal_diffusivity, h_thermal_diffusivity, size, hipMemcpyHostToDevice);

    printf("h_temp: %lf and d_temp: %lf \n", h_temp[0][3*(N+2) + 20], &d_temp[(3*(N+2) + 20)]);

}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( ! out ) {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    for ( int_t iter = 1; iter <= N; iter++)
    {
        fwrite( h_temp[0] + (M+2) * iter + 1, sizeof(real_t), N, out );
    }
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_temp[0] );
    free ( h_temp[1] );
    free ( h_thermal_diffusivity );

    // TODO 9: Free device memory.
    hipFree (d_temp);
    hipFree (d_temp_next);
    hipFree (d_thermal_diffusivity);
}
